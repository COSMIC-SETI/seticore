#include "hip/hip_runtime.h"
#include <assert.h>
#include "hipblas.h"
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <iostream>

#include "beamformer.h"
#include "cuda_util.h"
#include "util.h"

using namespace std;

const hipComplex COMPLEX_ONE = make_hipComplex(1.0, 0.0);
const hipComplex COMPLEX_ZERO = make_hipComplex(0.0, 0.0);

/*
  We convert from int8 input with format:
    input[block][antenna][coarse-channel][time-within-block][polarity][real or imag]

  to complex-float output with format:
    buffer[polarity][antenna][coarse-channel][time]

  block and time-within-block combine to form a single time index.
 */
__global__ void convertRaw(const int8_t* input, int input_size,
                           thrust::complex<float>* buffer, int buffer_size,
                           int nants, int nblocks, int num_coarse_channels, int npol, int nsamp,
                           int time_per_block) {
  int time_within_block = blockIdx.x * CUDA_MAX_THREADS + threadIdx.x;
  if (time_within_block >= time_per_block) {
    return;
  }
  int block = blockIdx.y;
  int antenna = blockIdx.z / num_coarse_channels;
  int chan = blockIdx.z % num_coarse_channels;
  int time = block * time_per_block + time_within_block;
  
  for (int pol = 0; pol < npol; ++pol) {
    int input_index = 2 * index5d(block, antenna, nants, chan, num_coarse_channels,
                                  time_within_block, time_per_block, pol, npol);
    int converted_index = index4d(pol, antenna, nants, chan, num_coarse_channels, time, nsamp);

    assert(input_index + 1 < input_size);
    assert(converted_index < buffer_size);
    
    buffer[converted_index] = thrust::complex<float>
      (input[input_index] * 1.0, input[input_index + 1] * 1.0);
  }
}

/*
  shift converts from the post-FFT format with format:
    buffer[polarity][antenna][coarse-channel][time][fine-channel]

  to a format ready for beamforming:
    prebeam[time][channel][polarity][antenna]

  We also toggle the high bit of the frequency fine channel. Hence "shift".
  This is like swapping the low half and the high half of the output of each FFT.
  It would be great for this comment to explain why this shift is necessary, but, I don't
  understand it myself, so I can't explain it.
 */
__global__ void shift(thrust::complex<float>* buffer, thrust::complex<float>* prebeam,
                      int fft_size, int nants, int npol, int num_coarse_channels,
                      int num_timesteps) {
  int antenna = threadIdx.y;
  int pol = threadIdx.z;
  int fine_chan = blockIdx.x;
  int coarse_chan = blockIdx.y;
  int time = blockIdx.z;

  int output_fine_chan = fine_chan ^ (fft_size >> 1);

  int input_index = index5d(pol, antenna, nants, coarse_chan, num_coarse_channels,
                            time, num_timesteps, fine_chan, fft_size);
  int output_index = index5d(time, coarse_chan, num_coarse_channels, output_fine_chan, fft_size,
                             pol, npol, antenna, nants);

  prebeam[output_index] = buffer[input_index];
}

/*
  Beamforming combines the channelized data with format:
    prebeam[time][coarse-channel][fine-channel][polarity][antenna]

  with the coefficient data, format:
    coefficients[coarse-channel][beam][polarity][antenna]

  to generate output beams with format:
    voltage[time][polarity][coarse-channel][fine-channel][beam]

  We combine prebeam with coefficients according to the indices they have in common,
  not conjugating the coefficients because we expect them to already be in the
  correct conjugation for multiplying, and then sum along antenna dimension to reduce.
*/
__global__ void beamform(const thrust::complex<float>* prebeam,
                         const thrust::complex<float>* coefficients,
                         thrust::complex<float>* voltage,
                         int fft_size, int nants, int nbeams, int num_coarse_channels,
                         int npol, int num_timesteps,
                         int prebeam_size, int voltage_size, int coefficients_size) {
  int antenna = threadIdx.x;
  int fine_chan = blockIdx.x;
  int coarse_chan = blockIdx.y;
  int beam = blockIdx.z;

  const int MAX_ANTS = 64;
  assert(nants <= MAX_ANTS);
  __shared__ thrust::complex<float> reduced[MAX_ANTS];

  for (int pol = 0; pol < npol; ++pol) {
    int coeff_index = index4d(coarse_chan, beam, nbeams, pol, npol, antenna, nants);
    assert(2 * coeff_index + 1 < coefficients_size);
    thrust::complex<float> conjugated = coefficients[coeff_index];
    for (int time = 0; time < num_timesteps; ++time) {
      int prebeam_index = index5d(time, coarse_chan, num_coarse_channels, fine_chan,
                                  fft_size, pol, npol, antenna, nants);
      assert(prebeam_index < prebeam_size);
      assert(antenna < MAX_ANTS);
      reduced[antenna] = prebeam[prebeam_index] * conjugated;

      __syncthreads();

      for (int k = MAX_ANTS / 2; k > 0; k >>= 1) {
        if (antenna < k && antenna + k < nants) {
          assert(antenna + k < MAX_ANTS);
          reduced[antenna] += reduced[antenna + k];
        }
        __syncthreads();
      }

      if (antenna == 0) {
        int voltage_index = index5d(time, pol, npol, coarse_chan, num_coarse_channels,
                                    fine_chan, fft_size, beam, nbeams);
        assert(voltage_index < voltage_size);
        voltage[voltage_index] = reduced[0];
      }
    }
  }
}

/*
  Runs beamforming just for the provided time and polarity, using cublas batch
  matrix multiplication.
  See the comment on the beamform kernel.

  This API is not immediately intuitive. See this blog post:
    https://developer.nvidia.com/blog/cublas-strided-batched-matrix-multiply/
  in particular their explanation of gemmStridedBatched.

  To convert into the notation used by the blog post:

  A = coefficients (which we will transpose)
  B = prebeam
  C = voltage
  m = beam
  n = fine channel
  p = coarse channel
  k = antenna
  alpha = 1.0
  beta = 0.0

  We are converting five-dimensional data plus four-dimensional data into
  five-dimensional output, so we fix time and polarity for each call to cublas.
  We could have fixed fine channel instead of time, or coarse channel instead of
  polarity, but it seems better to fix the smaller dimensions.
  Honestly, there are so many possibilities for how to arrange this data, I have
  not even begun to test all the ways it could work.
 */
void Beamformer::runCublasBeamform(int time, int pol) {
  // Calculate where the matrices start
  int coeff_offset = index4d(0, 0, nbeams, pol, npol, 0, nants);
  auto coeff_start = (const hipComplex*) (coefficients + coeff_offset);
  int prebeam_offset = index5d(time, 0, num_coarse_channels, 0, fft_size, pol, npol,
                               0, nants);
  auto prebeam_start = (const hipComplex*) (prebeam + prebeam_offset);
  int voltage_offset = index5d(time, pol, npol, 0, num_coarse_channels, 0, fft_size,
                               0, nbeams);
  auto voltage_start = (hipComplex*) (buffer + voltage_offset);

  // Calculate strides
  // ldA, the A-m stride (since we are transposing. normally it would be A-k)
  int coeff_beam_stride = index4d(0, 1, nbeams, 0, npol, 0, nants);
  // strideA, the A-p stride
  int coeff_coarse_stride = index4d(1, 0, nbeams, 0, npol, 0, nants);
  // ldB, the B-n stride
  int prebeam_fine_stride = index5d(0, 0, num_coarse_channels, 1, fft_size,
                                    0, npol, 0, nants);
  // strideB, the B-p stride
  int prebeam_coarse_stride = index5d(0, 1, num_coarse_channels, 0, fft_size,
                                      0, npol, 0, nants);
  // ldC, the C-n stride
  int voltage_fine_stride = index5d(0, 0, npol, 0, num_coarse_channels,
                                    1, fft_size, 0, nbeams);
  // strideC, the C-p stride
  int voltage_coarse_stride = index5d(0, 0, npol, 1, num_coarse_channels,
                                      0, fft_size, 0, nbeams);

  cublasCgemm3mStridedBatched
    (cublas_handle, 
     HIPBLAS_OP_T, HIPBLAS_OP_N,
     nbeams, fft_size, nants,
     &COMPLEX_ONE,
     coeff_start, coeff_beam_stride, coeff_coarse_stride, 
     prebeam_start, prebeam_fine_stride, prebeam_coarse_stride,
     &COMPLEX_ZERO,
     voltage_start, voltage_fine_stride, voltage_coarse_stride,
     num_coarse_channels);
  checkCuda("Beamformer runCublasBeamform");
}

/*
  We calculate power and shrink the data at the same time.
  Every polarity and every window of STI adjacent timesteps gets reduced to a single
  power value, by adding the norm of each complex voltage.

  The input voltages have format: 
    voltage[time][polarity][frequency][beam]

  and the output power has format:
    power[beam][time][frequency]

  where the time dimension has shrunk by a factor of STI, now indexed by [0, nwin).
  We add time_offset to all the times in the output.

  TODO: this also seems equivalent to a batch matrix multiplication. could we do this
  with a cublas routine?
 */
__global__ void calculatePower(const thrust::complex<float>* voltage,
                               float* power,
                               int nbeams, int num_channels, int npol,
                               int num_output_timesteps, int time_offset) {
  int chan = blockIdx.x;
  int beam = blockIdx.y;
  int coarse_timestep = blockIdx.z;
  int output_timestep = coarse_timestep + time_offset;
  
  int fine_timestep = threadIdx.x;
  assert(fine_timestep < STI);
  int time = coarse_timestep * STI + fine_timestep;

  assert(2 == npol);
  int pol0_index = index4d(time, 0, npol, chan, num_channels, beam, nbeams);
  int pol1_index = index4d(time, 1, npol, chan, num_channels, beam, nbeams);
  int power_index = index3d(beam, output_timestep, num_output_timesteps, chan, num_channels);

  __shared__ float reduced[STI];
  float real0 = voltage[pol0_index].real();
  float imag0 = voltage[pol0_index].imag();
  float real1 = voltage[pol1_index].real();
  float imag1 = voltage[pol1_index].imag();
  reduced[fine_timestep] = real0 * real0 + imag0 * imag0 + real1 * real1 + imag1 * imag1;

  __syncthreads();

  for (int k = STI / 2; k > 0; k >>= 1) {
    if (fine_timestep < k) {
      reduced[fine_timestep] += reduced[fine_timestep + k];
    }
    __syncthreads();
  }

  if (fine_timestep == 0) {
    power[power_index] = reduced[0];
  }
}

/*
  The Beamformer encapsulates the GPU memory allocations we use for beamforming.
  The workflow is to create a beamformer for a particular set of dimensions,
  use it to form many beams, and then destruct it when we want to free the memory.

  TODO: nants and npol are specified twice, once by the recipe file and once by the input.
  We should check to ensure they are the same and handle it cleanly if they aren't.
 */
Beamformer::Beamformer(int fft_size, int nants, int nbeams, int nblocks,
                       int num_coarse_channels, int npol, int nsamp)
  : fft_size(fft_size), nants(nants), nbeams(nbeams), nblocks(nblocks),
    num_coarse_channels(num_coarse_channels), npol(npol), nsamp(nsamp) {
  assert(0 == nsamp % (STI * fft_size));
  assert(0 == nsamp % nblocks);
  assert(roundUpToPowerOfTwo(fft_size) == fft_size);

  int frame_size = num_coarse_channels * nsamp;
  
  coefficients_size = 2 * nants * nbeams * num_coarse_channels * npol;
  size_t coefficients_bytes = coefficients_size * sizeof(float);
  hipMallocManaged(&coefficients, coefficients_bytes);
  checkCuda("Beamformer coefficients malloc");
 
  size_t fft_buffer_size = nants * npol * frame_size;
  size_t voltage_size = nbeams * npol * frame_size;
  buffer_size = max(fft_buffer_size, voltage_size);
  size_t buffer_bytes = buffer_size * sizeof(thrust::complex<float>);
  hipMallocManaged(&buffer, buffer_bytes);
  checkCuda("Beamformer buffer malloc");

  prebeam_size = nants * npol * frame_size;
  size_t prebeam_bytes = prebeam_size * sizeof(thrust::complex<float>);
  hipMallocManaged(&prebeam, prebeam_bytes);
  checkCuda("Beamformer prebeam malloc");

  power_size = nbeams * frame_size / STI;
  size_t power_bytes = power_size * sizeof(float);
  hipMallocManaged(&power, power_bytes);
  checkCuda("Beamformer power malloc");

  int batch_size = nants * npol;
  hipfftPlan1d(&plan, fft_size, HIPFFT_C2C, batch_size);
  checkCuda("Beamformer fft planning");

  hipblasCreate(&cublas_handle);
  checkCuda("Beamformer cublas handle");
  
  size_t total_bytes = coefficients_bytes + buffer_bytes + prebeam_bytes + power_bytes;
  cout << "beamformer memory: " << prettyBytes(total_bytes) << endl;
}

Beamformer::~Beamformer() {
  hipFree(coefficients);
  hipFree(buffer);
  hipFree(prebeam);
  hipFree(power);
  hipfftDestroy(plan);
  hipblasDestroy(cublas_handle);
}

int Beamformer::numOutputChannels() const {
  return num_coarse_channels * fft_size;
}

int Beamformer::numOutputTimesteps() const {
  return nsamp / (fft_size * STI); 
}

/*
  Power from beamforming the input is written into output, with an offset
  of time_offset.

  The format of the input is row-major:
    input[block][antenna][coarse-channel][time-within-block][polarity][real or imag]

  The format of the output is row-major:
     power[beam][time][channel]
  but its time resolution has been reduced by a factor of (fft_size * STI).
 */
void Beamformer::run(RawBuffer& input, MultibeamBuffer& output, int time_offset) {
  int time_per_block = nsamp / nblocks;
  // Unfortunate overuse of "block"
  int cuda_blocks_per_block = (time_per_block + CUDA_MAX_THREADS - 1) / CUDA_MAX_THREADS;
  dim3 convert_raw_block(CUDA_MAX_THREADS, 1, 1);
  dim3 convert_raw_grid(cuda_blocks_per_block, nblocks, nants * num_coarse_channels);
  convertRaw<<<convert_raw_grid, convert_raw_block>>>
    (input.data, input.data_size,
     buffer, buffer_size,
     nants, nblocks, num_coarse_channels, npol, nsamp, time_per_block);
  checkCuda("Beamformer convertRaw");

  // Run FFTs. TODO: see if there's a faster way
  int num_ffts = nants * npol * num_coarse_channels * nsamp / fft_size;
  int batch_size = nants * npol;
  int num_batches = num_ffts / batch_size;
  for (int i = 0; i < num_batches; ++i) {
    hipComplex* pointer = (hipComplex*) buffer + i * batch_size * fft_size;
    hipfftExecC2C(plan, pointer, pointer, HIPFFT_FORWARD);
  }
  checkCuda("Beamformer fft operation");

  dim3 shift_block(1, nants, npol);
  dim3 shift_grid(fft_size, num_coarse_channels, nsamp / fft_size);
  shift<<<shift_grid, shift_block>>>(buffer, prebeam, fft_size, nants, npol,
                                     num_coarse_channels, nsamp / fft_size);
  checkCuda("Beamformer shift");
  
  dim3 beamform_block(nants, 1, 1);
  dim3 beamform_grid(fft_size, num_coarse_channels, nbeams);
  beamform<<<beamform_grid, beamform_block>>>(prebeam, coefficients, buffer, fft_size,
                                              nants, nbeams, num_coarse_channels, npol,
                                              nsamp / fft_size, prebeam_size, buffer_size,
                                              coefficients_size);

  dim3 power_block(STI, 1, 1);
  dim3 power_grid(numOutputChannels(), nbeams, numOutputTimesteps());
  calculatePower<<<power_grid, power_block>>>
    (buffer, output.data, nbeams, numOutputChannels(), npol, numOutputTimesteps(),
     time_offset);
  checkCuda("Beamformer calculatePower");
}

thrust::complex<float> Beamformer::getCoefficient(int antenna, int pol, int beam,
                                                  int coarse_channel) const {
  hipDeviceSynchronize();
  checkCuda("Beamformer getCoefficient");
  assert(antenna < nants);
  assert(pol < npol);
  assert(beam < nbeams);
  assert(coarse_channel < num_coarse_channels);
  int i = index4d(coarse_channel, beam, nbeams, pol, npol, antenna, nants);
  return coefficients[i];
}

// The last index can be either time's fine index or the fine channel index, depending
// on whether it's pre-FFT or post-FFT.
thrust::complex<float> Beamformer::getFFTBuffer(int pol, int antenna, int coarse_channel,
                                                int time, int last_index) const {
  hipDeviceSynchronize();
  checkCuda("Beamformer getFFTBuffer");
  assert(pol < npol);
  assert(antenna < nants);
  assert(coarse_channel < num_coarse_channels);
  assert(time * fft_size < nsamp);
  assert(last_index < fft_size);
  int i = index5d(pol, antenna, nants, coarse_channel, num_coarse_channels,
                  time, nsamp / fft_size, last_index, fft_size);
  return buffer[i];
}

thrust::complex<float> Beamformer::getPrebeam(int time, int channel, int pol, int antenna) const {
  hipDeviceSynchronize();
  checkCuda("Beamformer getPrebeam");
  assert(time < nsamp);
  assert(channel < num_coarse_channels * fft_size);
  assert(pol < npol);
  assert(antenna < nants);
  int i = index4d(time, channel, num_coarse_channels * fft_size, pol, npol, antenna, nants);
  return prebeam[i];
}

thrust::complex<float> Beamformer::getVoltage(int time, int pol, int channel, int beam) const {
  hipDeviceSynchronize();
  checkCuda("Beamformer getVoltage");
  assert(time < nsamp);
  assert(pol < npol);
  assert(channel < numOutputChannels());
  assert(beam < nbeams);
  int i = index4d(time, pol, npol, channel, numOutputChannels(), beam, nbeams);
  return buffer[i];
}


#include "catch/catch.hpp"
#include <fmt/core.h>
#include <iostream>

#include "filterbank_buffer.h"
#include "taylor.h"

TEST_CASE("tiled taylor outputs match basic algorithm", "[taylor]") {
  for (int num_timesteps = 4; num_timesteps <= 32; num_timesteps *= 2) {
    int num_channels = 2000;
    FilterbankBuffer input(num_timesteps, num_channels);
    for (int time = 0; time < num_timesteps; ++time) {
      for (int chan = 0; chan < num_channels; ++chan) {
        input.set(time, chan, 1.0 * (1 + time + chan)); 
      }
    }
  
    FilterbankBuffer buffer1(num_timesteps, num_channels);
    FilterbankBuffer buffer2(num_timesteps, num_channels);
    FilterbankBuffer tiled(num_timesteps, num_channels);

    for (int drift_block = -2; drift_block <= 2; ++drift_block) {
      // Avoid confusion while debugging
      buffer1.zero();
      buffer2.zero();
      tiled.zero();
      hipDeviceSynchronize();
    
      const float* out_ptr = basicTaylorTree(input.data, buffer1.data, buffer2.data,
                                             num_timesteps, num_channels, drift_block);
      const FilterbankBuffer& basic = (out_ptr == buffer1.data) ? buffer1 : buffer2;

      tiledTaylorTree(input.data, tiled.data, num_timesteps, num_channels, drift_block);

      hipDeviceSynchronize();

      basic.assertEqual(tiled, drift_block);
    }
  }
}

TEST_CASE("two-pass taylor outputs match basic algorithm", "[taylor]") {
  int num_timesteps = 64;
  int num_channels = 1000;
  FilterbankBuffer input(num_timesteps, num_channels);
  for (int time = 0; time < num_timesteps; ++time) {
    for (int chan = 0; chan < num_channels; ++chan) {
      input.set(time, chan, 1.0 * (1 + time + chan)); 
    }
  }
  
  FilterbankBuffer buffer1(num_timesteps, num_channels);
  FilterbankBuffer buffer2(num_timesteps, num_channels);

  FilterbankBuffer two_pass_buffer(num_timesteps, num_channels);
  FilterbankBuffer two_pass_output(num_timesteps, num_channels);

  for (int drift_block = 0; drift_block <= 2; ++drift_block) {
    // Avoid confusion while debugging
    buffer1.zero();
    buffer2.zero();
    two_pass_buffer.zero();
    two_pass_output.zero();
    hipDeviceSynchronize();
    
    const float* out_ptr = basicTaylorTree(input.data, buffer1.data, buffer2.data,
                                           num_timesteps, num_channels, drift_block);
    const FilterbankBuffer& basic = (out_ptr == buffer1.data) ? buffer1 : buffer2;
    // const FilterbankBuffer& other = (out_ptr == buffer1.data) ? buffer2 : buffer1;

    twoPassTaylorTree(input.data, two_pass_buffer.data, two_pass_output.data,
                      num_timesteps, num_channels, drift_block);

    hipDeviceSynchronize();

    basic.assertEqual(two_pass_output, drift_block);
  }
  
}

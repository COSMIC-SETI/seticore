#include "multibeam_buffer.h"

#include <assert.h>
#include "cuda_util.h"

using namespace std;

MultibeamBuffer::MultibeamBuffer(int num_beams, int num_timesteps, int num_channels)
  : num_beams(num_beams), num_timesteps(num_timesteps), num_channels(num_channels) {
  hipMallocManaged(&data, sizeof(float) * num_beams * num_timesteps * num_channels);
  checkCuda("MultibeamBuffer data malloc");
}

MultibeamBuffer::~MultibeamBuffer() {
  hipFree(data);
}

FilterbankBuffer MultibeamBuffer::getBeam(int beam) {
  assert(0 <= beam && beam < num_beams);
  int beam_size = num_timesteps * num_channels;
  return FilterbankBuffer(num_timesteps, num_channels, data + beam * beam_size);
}

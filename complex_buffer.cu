#include <assert.h>
#include "complex_buffer.h"
#include "cuda_util.h"

using namespace std;

ComplexBuffer::ComplexBuffer(size_t size) :
  size(size), bytes(sizeof(thrust::complex<float>) * size) {

  hipMallocManaged(&data, bytes);
  checkCudaMalloc("ComplexBuffer", bytes);
}

ComplexBuffer::~ComplexBuffer() {
  hipFree(data);
}

thrust::complex<float> ComplexBuffer::get(int index) const {
  assert(index >= 0);
  assert(index < size);
  hipDeviceSynchronize();
  return data[index];
}

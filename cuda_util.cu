#include <iostream>

#include "cuda_util.h"

using namespace std;

void checkCuda(const string& tag) {
  auto err = hipGetLastError();
  if (err != hipSuccess) {
    cerr << tag << ": cuda error " << err << ": " << hipGetErrorString(err) << endl;
    exit(1);
  }
}

Stream::Stream() {
  hipStreamCreate(&stream);
  checkCuda("hipStreamCreate");
}

Stream::~Stream() {
  hipStreamDestroy(stream);
  checkCuda("hipStreamDestroy");
}

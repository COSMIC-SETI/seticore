#include <iostream>

#include "cuda_util.h"

using namespace std;

void checkCuda(const string& tag) {
  auto err = hipGetLastError();
  if (err != hipSuccess) {
    cerr << tag << ": cuda error " << err << ": " << hipGetErrorString(err) << endl;
    exit(2);
  }
}

void checkCudaMalloc(const string& tag, size_t bytes) {
  auto err = hipGetLastError();
  if (err != hipSuccess) {
    cerr << tag << ": cuda error " << err << " allocating " <<
      bytes << " bytes: " << hipGetErrorString(err) << endl;
    exit(2);
  }
}

Stream::Stream() {
  hipStreamCreate(&stream);
  checkCuda("hipStreamCreate");
}

Stream::~Stream() {
  hipStreamDestroy(stream);
  checkCuda("hipStreamDestroy");
}

#include <iostream>

#include "cuda_util.h"

using namespace std;

void checkCuda(hipError_t err) {
  if (err != 0) {
    cerr << "cuda error " << err << ": " << hipGetErrorString(err) << endl;
    exit(1);
  }
}


#include <assert.h>
#include "cuda_util.h"
#include "multiantenna_buffer.h"
#include <iostream>

MultiantennaBuffer::MultiantennaBuffer(int num_timesteps, int num_channels,
                                       int num_polarities, int num_antennas)
  : ComplexBuffer((size_t) num_timesteps * num_channels * num_polarities * num_antennas),
    num_timesteps(num_timesteps), num_channels(num_channels),
    num_polarities(num_polarities), num_antennas(num_antennas) {
}

thrust::complex<float> MultiantennaBuffer::get(int time, int channel,
                                               int polarity, int antenna) const {
  assert(0 <= time && time < num_timesteps);
  assert(0 <= channel && channel < num_channels);
  assert(0 <= polarity && polarity < num_polarities);
  assert(0 <= antenna && antenna < num_antennas);
  int index = index4d(time, channel, num_channels, polarity, num_polarities,
                      antenna, num_antennas);
  return get(index);
}
                                       
void MultiantennaBuffer::copyRange(int src_start_channel,
                                   MultiantennaBuffer& dest, int dest_start_time) const {
  assert(src_start_channel >= 0);
  assert(src_start_channel + dest.num_channels <= num_channels);
  assert(dest_start_time >= 0);
  assert(dest_start_time + num_timesteps <= dest.num_timesteps);
  assert(num_polarities == dest.num_polarities);
  assert(num_antennas == dest.num_antennas);

  int src_index = index4d(0, src_start_channel, num_channels,
                             0, num_polarities, 0, num_antennas);
  int dest_index = index4d(dest_start_time, 0, dest.num_channels,
                           0, num_polarities, 0, num_antennas);

  size_t entry_size = sizeof(thrust::complex<float>) * num_polarities * num_antennas;
  size_t src_pitch = entry_size * num_channels;
  size_t dest_pitch = entry_size * dest.num_channels;

  auto src_ptr = data + src_index;
  auto dest_ptr = dest.data + dest_index;
  
  hipMemcpy2DAsync(dest_ptr, dest_pitch,
                    src_ptr, src_pitch,
                    dest_pitch, num_timesteps,
                    hipMemcpyDefault);
  checkCuda("MultiantennaBuffer copyRange");
}

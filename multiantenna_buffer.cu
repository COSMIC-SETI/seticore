#include <assert.h>
#include "cuda_util.h"
#include "multiantenna_buffer.h"
#include <iostream>

MultiantennaBuffer::MultiantennaBuffer(int num_timesteps, int num_channels,
                                       int num_polarity, int num_antennas)
  : ComplexBuffer((size_t) num_timesteps * num_channels * num_polarity * num_antennas),
    num_timesteps(num_timesteps), num_channels(num_channels),
    num_polarity(num_polarity), num_antennas(num_antennas) {
}

thrust::complex<float> MultiantennaBuffer::get(int time, int channel,
                                               int polarity, int antenna) const {
  assert(0 <= time && time < num_timesteps);
  assert(0 <= channel && channel < num_channels);
  assert(0 <= polarity && polarity < num_polarity);
  assert(0 <= antenna && antenna < num_antennas);
  int index = index4d(time, channel, num_channels, polarity, num_polarity,
                      antenna, num_antennas);
  return get(index);
}
                                       
void MultiantennaBuffer::copyRange(int src_start_channel,
                                   MultiantennaBuffer& dest, int dest_start_time) const {
  assert(src_start_channel >= 0);
  assert(src_start_channel + dest.num_channels <= num_channels);
  assert(dest_start_time >= 0);
  assert(dest_start_time + num_timesteps <= dest.num_timesteps);
  assert(num_polarity == dest.num_polarity);
  assert(num_antennas == dest.num_antennas);

  int src_index = index4d(0, src_start_channel, num_channels,
                             0, num_polarity, 0, num_antennas);
  int dest_index = index4d(dest_start_time, 0, dest.num_channels,
                           0, num_polarity, 0, num_antennas);

  size_t entry_size = sizeof(thrust::complex<float>) * num_polarity * num_antennas;
  size_t src_pitch = entry_size * num_channels;
  size_t dest_pitch = entry_size * dest.num_channels;

  auto src_ptr = data + src_index;
  auto dest_ptr = dest.data + dest_index;
  
  hipMemcpy2DAsync(dest_ptr, dest_pitch,
                    src_ptr, src_pitch,
                    dest.num_channels, num_timesteps,
                    hipMemcpyDefault);
  checkCuda("MultiantennaBuffer copyRange");
}

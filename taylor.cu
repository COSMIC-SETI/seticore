#include "hip/hip_runtime.h"
#include <assert.h>
#include <hip/hip_runtime.h>
#include <iostream>

#include "cuda_util.h"
#include "taylor.h"

using namespace std;

/*
  Apologies for the length of this comment, but the Taylor tree algorithm is
  fairly complicated for the number of lines of code it is, so it takes
  a while to explain. My hope is that this code will be comprehensible
  for people that have not seen the Taylor tree algorithm before.

  These paths are diagonal paths through the data, touching one element
  per row, using the nearest cell to a straight line. For example, if
  num_timesteps is 4, the paths for each path_offset look like:

  path_offset:   0        1         2          3
                 ..X..    ..X...    ..X....    ..X.....
                 ..X..    ..X...    ...X...    ...X....
                 ..X..    ...X..    ...X...    ....X...
                 ..X..    ...X..    ....X..    .....X..

  At a high level, you can recursively calculate all the sums of these
  paths in O(n log n) operations by running on the top half, then the
  bottom half, then adding them up appropriately.

  The key to understanding this code is to understand the format of
  the buffers: source_buffer and target_buffer.
  source_buffer and target_buffer store the sum along an
  approximately-linear path through the input data. The best way to
  think of them is as three-dimensional arrays, indexed like

  buffer[time_block][path_offset][start_frequency]

  path_offset is the difference between the frequency of the starting
  point of the path and the ending point of the path. It's in the range:
  [0, path_length)

  start_frequency is the index of the starting frequency, in the range:
  [0, num_freqs)

  time_block is a bit weirder. In our recursion, we don't need to keep
  sums for every possible start time. We can cut the number of start
  times in half, every step through the recursion. After n steps of
  recursion, we only need to keep a sum for every 2^n timesteps. So if
  start_time is the index of the starting time, in [0, num_timesteps),
  time_block obeys the relation

  time_block * path_length = start_time

  time_block thus is in the range:
  [0, num_timesteps / path_length)

  and each time block represents data for sums over path_length
  different start times.

  So each buffer holds (num_timesteps * num_freqs) elements.

  When we read input data, it's normally thought of as a two-dimensional
  array, indexed like

  input[time][frequency]

  Since we just pass the buffers around as one-dimensional arrays, this
  is essentially equivalent to thinking of it as a three-dimensional
  array in the above format, with path_offset always equal to zero.

  When we finish running the Taylor tree algorithm, time_block will
  always be zero. Thus we can think of the final output as a
  two-dimensional array as well, indexed like

  output[path_offset][start_frequency]

  It's really just for understanding the intervening steps that it's
  better to think of these buffers as being three-dimensional arrays.

  There's one more detail: drift blocks. So far we've explained the case
  where drift_block = 0, and we are calculating slopes between vertical
  and one horizontal-step-per-vertical step. You can think of this as
  the drift range [0, 1] when measured in units of
  horizontal-step-per-vertical-step. We can use a similar algorithm to
  calculate sums for all slopes in [drift_block, drift_block+1], if we just shift all
  accesses of the kth timestep by an extra drift_block * k.
*/

/*
  Kernel to run one round of the Taylor tree algorithm on an input array.

  We assume that the caller is using a grid tiling such that
  blockIdx.x * blockDim.x + threadIdx.x
  will cover all frequencies.
*/
__global__ void taylorTreeOneStepKernel(const float* source_buffer, float* target_buffer,
                                        int num_timesteps, int num_freqs, int path_length,
                                        int drift_block) {
  assert(path_length <= num_timesteps);
  int freq = blockIdx.x * blockDim.x + threadIdx.x;
  if (freq < 0 || freq >= num_freqs) {
    return;
  }

  taylorOneStepOneChannel(source_buffer, target_buffer,
                          freq, num_timesteps, num_freqs, num_freqs, path_length,
                          drift_block);
}

/*
  Run all rounds of the Taylor tree algorithm.
  buffer1 and buffer2 are two GPU buffers provided to do work.
  Returns the buffer that the eventual output is in.
 */
const float* fullTaylorTree(const float* input, float* buffer1, float* buffer2,
                            int num_timesteps, int num_channels, int drift_block) {
  // This will create one cuda thread per frequency bin
  int grid_size = (num_channels + CUDA_MAX_THREADS - 1) / CUDA_MAX_THREADS;

  // The dataflow among the buffers looks like:
  // input -> buffer1 -> buffer2 -> buffer1 -> buffer2 -> ...
  // We use the aliases source_buffer and target_buffer to make this simpler.
  // In each pass through the upcoming loop, we are reading from
  // source_buffer and writing to target_buffer.
  const float* source_buffer = input;
  float* target_buffer = buffer1;

  // Each pass through the data calculates the sum of paths that are
  // twice as long as the previous path, until we reach our goal,
  // which is paths of length num_timesteps.
  for (int path_length = 2; path_length <= num_timesteps; path_length *= 2) {

    // Invoke cuda kernel
    taylorTreeOneStepKernel<<<grid_size, CUDA_MAX_THREADS>>>
      (source_buffer, target_buffer, num_timesteps, num_channels,
       path_length, drift_block);
    checkCuda("taylorTreeOneStepKernel");

    // Swap buffer aliases to make the old target the new source
    if (target_buffer == buffer1) {
      source_buffer = buffer1;
      target_buffer = buffer2;
    } else if (target_buffer == buffer2) {
      source_buffer = buffer2;
      target_buffer = buffer1;
    } else {
      cerr << "programmer error; control flow should not reach here\n";
      exit(1);
    }
  }

  // The final path sums are in source_buffer because we did one last
  // alias-swap
  return source_buffer;
}
